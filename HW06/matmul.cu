#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef MATMUL_CUH

// Computes the matrix product of A and B, storing the result in C.
// Each thread should compute _one_ element of output.
// Does not use shared memory for this problem.
//
// A, B, and C are row major representations of nxn matrices in device memory.
//
// Assumptions:
// - 1D kernel configuration
__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n){
    // threadIdx = i
    // blockIdx = j
    // need to iterate k

    int iIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (iIndex >= n*n) return;

    for (int k = 0; k < n; k++){
        int jIndex = (iIndex / n) * n + k;
        int kIndex = k * n + (iIndex % n);
        if (k == 0) printf("INDEX 0 -> index = %d | j = %d | k = %d\n", iIndex, jIndex, kIndex);
        if (k == 1) printf("INDEX 1 -> index = %d | j = %d | k = %d\n", iIndex, jIndex, kIndex);
        C[iIndex] += A[jIndex] * B[kIndex];
    }
}

// Makes one call to matmul_kernel with threads_per_block threads per block.
// You can consider following the kernel call with cudaDeviceSynchronize (but if you use 
// cudaEventSynchronize to time it, that call serves the same purpose as cudaDeviceSynchronize).
void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){
    int blocks = (n*n + threads_per_block - 1) / threads_per_block;
    matmul_kernel<<<blocks, threads_per_block>>>(A, B, C, n);
    hipDeviceSynchronize();
}

#endif