#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "reduce.cuh"
using namespace std;

int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);
    int t = atoi(argv[2]);
    int blocks = (n + t - 1) / t / 2;
    
    int hInput[n], hOutput[blocks], *dInput, *dOutput;

    // randomization:
    srand(chrono::system_clock::now().time_since_epoch().count());

    // array initialization:
    for (int i = 0; i < n; i++){
        hInput[i] = static_cast <int> (rand() / static_cast <int> (RAND_MAX / 2)) - 1;
    }

    auto start = chrono::steady_clock::now();

    hipMalloc((void**)&dInput, sizeof(int) * n);
    hipMalloc((void**)&dOutput, sizeof(int) * blocks);
    hipMemcpy(dInput, &hInput, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemset(dOutput, 0, sizeof(int) * blocks);

    // do math:
    reduce(&dInput, &dOutput, n, t);

    // results:
    hipMemcpy(&hOutput, dOutput, sizeof(int) * blocks, hipMemcpyDeviceToHost);

    auto end = chrono::steady_clock::now();
	auto timePassed = chrono::duration_cast<std::chrono::microseconds>(end - start);

    cout << "element count:  \t" << n << endl;
	cout << "time to process:\t" << (timePassed.count() / 1000) << " milliseconds\n";
	cout << "result:         \t" << hOutput[0] << endl;

    hipFree(dInput);
    hipFree(dOutput);

    return 0;
}