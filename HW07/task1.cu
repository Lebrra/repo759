#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "matmul.cuh"
using namespace std;

template <typename T>
void doMatmul(int n, int blockSize){
    T hB[n*n], hA[n*n], hC[n*n], *dB, *dA, *dC;

    // randomization:
    srand(chrono::system_clock::now().time_since_epoch().count());

    // array initialization:
    for (int i = 0; i < n*n; i++){
        hA[i] = static_cast <T> (rand() / static_cast <T> (RAND_MAX / 20)) - 10;
        hB[i] = static_cast <T> (rand() / static_cast <T> (RAND_MAX / 20)) - 10;
    }

    auto start = chrono::steady_clock::now();

    hipMalloc((void**)&dA, sizeof(T) * n * n);
    hipMalloc((void**)&dB, sizeof(T) * n * n);
    hipMalloc((void**)&dC, sizeof(T) * n * n);
    hipMemcpy(dA, &hA, sizeof(T) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(dB, &hB, sizeof(T) * n * n, hipMemcpyHostToDevice);
    hipMemset(dC, 0, n * n * sizeof(T));

    // do math:
    cout << "Switch time\n";
    switch(sizeof(T)){
        case sizeof(int):
            cout << "Calculating type: int" << endl;
            matmul_1(dA, dB, dC, n, blockSize);
            break;
        case sizeof(float):
            cout << "Calculating type: float" << endl;
            matmul_2(dA, dB, dC, n, blockSize);
            break;
        case sizeof(double):
            cout << "Calculating type: double" << endl;
            matmul_3(dA, dB, dC, n, blockSize);
            break;
        default:
            cout << "Invalid type to process matmul.\n";
            return;
    }

    // results:
    hipMemcpy(&hC, dC, sizeof(T) * n * n, hipMemcpyDeviceToHost);

    auto end = chrono::steady_clock::now();
	auto timePassed = chrono::duration_cast<std::chrono::microseconds>(end - start);

    //cout << "Results of type " << typeof(T) << ":" << endl;
    cout << "element count:  \t" << n << endl;
	cout << "time to process:\t" << (timePassed.count() / 1000) << " milliseconds\n";
	cout << "first element:  \t" << hC[0] << endl;
	cout << "last element:   \t" << hC[n*n - 1] << endl << endl;

    hipFree(dB);
    hipFree(dA);
    hipFree(dC);
}

int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);
    int block = atoi(argv[2]);
    int type = atoi(argv[3]);
    
    cout << "I should execute matmul_" << type << " with an " << n << "x" << n << " matrix and " << block << " blocks\n";

    return 0;
}