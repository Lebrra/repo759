#include <hip/hip_runtime.h>
#include <iostream> 
using namespace std; 

#ifndef MATMUL_CUH

// (the difference is types of data)

template <typename T>
__global__ void matmul(const T *A, const T *B, T *C, unsigned int n, unsigned int block_dim){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int c = n * (block_dim * by + ty) + (block_dim * bx + tx);
    if (c >= n*n) return;

    int aStart = n * block_dim * by;
    int aEnd = aStart + n - 1;
    int aStep = block_dim;

    int bStart = block_dim * bx;
    int bStep = block_dim * n;

    T cSub = 0;
    
    extern __shared__ char shared[];
    T* As = (T*)shared;
    T* Bs = (T*)&As[block_dim*block_dim];  

    for (int a = aStart, b = bStart; a <= aEnd; a += aStep, b += bStep){
        As[ty * block_dim + tx] = A[a + n * ty + tx];
        Bs[ty * block_dim + tx] = B[b + n * ty + tx];
        __syncthreads();

        for (int k = 0; k < block_dim; k++){
            cSub += As[ty * block_dim + k] * Bs[k * block_dim + tx];
        }
            
        __syncthreads();
    }

    C[c] = cSub;
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul<int><<<dimGrid, dimBlock, block_dim*block_dim*2 * sizeof(int)>>>(A, B, C, n, block_dim);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul<float><<<dimGrid, dimBlock, block_dim*block_dim*2 * sizeof(float)>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}
__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + block_dim - 1) / block_dim, (n + block_dim - 1) / block_dim);
    matmul<double><<<dimGrid, dimBlock, block_dim*block_dim*2 * sizeof(double)>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}

#endif