#include <hip/hip_runtime.h>
#include <iostream> 
using namespace std; 

#ifndef MATMUL_CUH

// (the difference is types of data)

//template <typename T>
__global__ void matmul(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int c = n * block_dim * by + block_dim * bx;
    if (c >= n*n) return;

    int aStart = n * block_dim * by;
    int aEnd = aStart + n - 1;
    int aStep = block_dim;

    int bStart = block_dim * bx;
    int bStep = block_dim * n;

    int cSub = 0;
    
    __shared__ int As[1000][1000];
    __shared__ int Bs[1000][1000];

    for (int a = aStart, b = bStart; a <= aEnd; a += aStep, b += bStep){
        As[ty][tx] = A[a + n * ty + tx];
        Bs[ty][tx] = B[b + n * ty + tx];
        __syncthreads();

        for (int k = 0; k < block_dim; k++)
            cSub += As[ty][k] * Bs[k][tx];
        __syncthreads();
    }

    C[c + n * ty + tx] = cSub;
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(n/dimBlock.x, n/dimBlock.y);
    matmul<<<dimGrid, dimBlock>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(n/dimBlock.x, n/dimBlock.y);
    //matmul<float><<dimGrid, dimBlock>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}
__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim){
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(n/dimBlock.x, n/dimBlock.y);
    //matmul<double>l<<dimGrid, dimBlock>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}

#endif