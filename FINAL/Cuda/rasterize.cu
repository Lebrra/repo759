#include "hip/hip_runtime.h"
// inspiration: https://github.com/ssloy/tinyrenderer/wiki/Lesson-2:-Triangle-rasterization-and-back-face-culling

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "pixel.cuh"
//#include "sizeAdjuster.cuh"
#include "filehandler.cuh"

using namespace std;

// hardset output height and width
const int definedSize = 256;
const float padding = 10;

__global__ void adjustValue(float* vertices, int vertexCount, float minX, float minY, float padding, float multiplier){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Printing in kernel %d", index);
    if (index >= vertexCount || index % 3 == 2) return;
    // ignore z for now its not being used

    if (index % 3 == 0 && minX < 0){
        vertices[index] -= minX;
    }
    else if (index % 3 == 1 && minY < 0){
        vertices[index] -= minY; 
    }

    vertices[index] *= multiplier;
    vertices[index] += padding;
}

__host__ void adjustSize(float* vertices, int vertexCount, float size, float padding){
    printf("within adjustSize...\n");
    float minX = 0;
    float maxX = 0;
    float minY = 0;
    float maxY = 0;

    // calculate min and max -es
    for (int i = 0; i < vertexCount; i++) {
        int x = i * 3;  // x + 1 = y

        if (vertices[x] < minX) minX = vertices[x];
        if (vertices[x] > maxX) maxX = vertices[x];
        if (vertices[x + 1] < minY) minY = vertices[x + 1];
        if (vertices[x + 1] > maxY) maxY = vertices[x + 1];
    }
    printf("completed min/max...\n");

    // create multiplier based off larger difference
    float pointsWidth = maxX - minX;
    float pointsHeight = maxY - minY;

    float multiplier;
    if (pointsWidth > pointsHeight) {
        multiplier = (size - padding*2) / pointsWidth;
    }
    else { 
        multiplier = (size - padding*2) / pointsHeight;
    }

    printf("ready to calculate blocks...\n");

    // apply multiplier to all points (and offset if any points are negative)
    int blocks = ((vertexCount*3) + 256 - 1) / 256;
    printf("applying adjustments using block count: %d\n", blocks);
    adjustValue<<<blocks, 256>>>(vertices, vertexCount, minX, minY, padding, multiplier);
    hipDeviceSynchronize();
}

int main(int argc, char** argv) {
    if (argc <= 1){
        cout << "Please include a filename (excluding the '_vertices.txt' or '_faces.txt') to rasterize!";
        return 0;
    }
    // else I could add optional size values, but let's just hardset it to 256...
    if (definedSize > 1024 || definedSize <= 0){
        cout << "Invalid image size; please pick a value between 1 and 1024";
        return 0;
    }

    auto start = chrono::steady_clock::now();

    // seeding with chrono so I don't have to include the ctime library:
	srand(chrono::system_clock::now().time_since_epoch().count());

    string fileName = argv[1];
    cout << "Processing file: " << fileName << endl;

    // gather data:
    int vertCount = getVertexCount(fileName);
    int triangleCount = getFaceCount(fileName);

    float* vertices = (float*)malloc(sizeof(float) * vertCount * 3);
    int* faces = (int*)malloc(sizeof(int) * triangleCount * 3);
    //float vertices[vertCount*3]; 
    //int faces[triangleCount*3];
    //for(int i = 0; i < max(vertCount, triangleCount)*3; i++){
    //    if (i < vertCount*3) vertices[i] = i;
    //    if (i < triangleCount*3) faces[i] = i;
    //}
    float *dVerts;
    
    readVertices(fileName, vertices);
    readFaces(fileName, faces);

    cout << "This file has " << vertCount << " vertices and " << triangleCount << " triangles!" << endl;

    cout << "Adjusting to size..." << endl;
    hipMalloc((void**)&dVerts, sizeof(float) * vertCount*3);
    hipMemcpy(dVerts, &vertices, sizeof(float) * vertCount*3, hipMemcpyHostToDevice);

    cout << "Calling adjust..." << endl;
    adjustSize(dVerts, vertCount, definedSize, padding);
    cout << "Completed adjusting..." << endl;
    hipMemcpy(&vertices, dVerts, sizeof(float) * vertCount*3, hipMemcpyDeviceToHost);
    hipFree(dVerts);

    int pointTests[definedSize * definedSize], *dPoints;
    hipMalloc((void**)&dPoints, sizeof(int) * definedSize * definedSize);
    hipMemset(dPoints, 0, sizeof(int) * definedSize * definedSize);

    // do math
    int validTriangles = 0;
    cout << "Comparing pixels with triangles..." << endl;
    float triangle[6], *dTri;
    hipMalloc((void**)&dTri, sizeof(float) * 6);
    for(int tri = 0; tri < triangleCount; tri++){
        int face1 = faces[tri * 3];
        int face2 = faces[tri * 3 + 1];
        int face3 = faces[tri * 3 + 2];

        triangle[0] = vertices[face1*3];
        triangle[1] = vertices[face1*3 + 1];
        triangle[2] = vertices[face2*3];
        triangle[3] = vertices[face2*3 + 1];
        triangle[4] = vertices[face3*3];
        triangle[5] = vertices[face3*3 + 1];

        // validate triangle: (if two points are only different in the z direction then let's just skip it)
        if ((triangle[0] == triangle[2] && triangle[1] == triangle[3]) ||
            (triangle[0] == triangle[4] && triangle[1] == triangle[5]) ||
            (triangle[2] == triangle[4] && triangle[3] == triangle[5])){
                continue;
            }
        validTriangles++;
        hipMemcpy(dTri, &triangle, sizeof(float) * 6, hipMemcpyHostToDevice);

        // do parallelism here
        inTriangle<<<definedSize, definedSize>>>(dTri, dPoints, validTriangles);
    }
    hipFree(dTri);

    hipMemcpy(&pointTests, dPoints, sizeof(int) * definedSize * definedSize, hipMemcpyDeviceToHost);
    hipFree(dPoints);

    // create random colors:
    cout << "Generating triangle colors...\n";
    float* colors = (float*)malloc(sizeof(float) * 3 * validTriangles);
    for(int i = 0; i < validTriangles * 3; i++){
        colors[i] = static_cast <float> (rand() / static_cast <float> (RAND_MAX));
    }

    auto end = chrono::steady_clock::now();
	auto timePassed = chrono::duration_cast<std::chrono::microseconds>(end - start);

    // write
    cout << "Rasterization completed after " << (timePassed.count() / 1000) << "ms!\nWriting results...\n";
    float* pixel = (float*)malloc(sizeof(float) * 3);
    readyOutputFile(fileName, (timePassed.count() / 1000));
    for (int i = 0; i < definedSize * definedSize; i++){
        if (pointTests[i] <= 0) {
            pixel[0] = pixel[1] = pixel[2] = 0;
        }
        else {
            // do color
            pixel[0] = colors[(pointTests[i] - 1)*3];
            pixel[1] = colors[(pointTests[i] - 1)*3 + 1];
            pixel[2] = colors[(pointTests[i] - 1)*3 + 2];
        }
        writeVertex(fileName, pixel);
    }
    free(pixel);

    cout << "All done!" << endl;
    auto realEnd = chrono::steady_clock::now();
	timePassed = chrono::duration_cast<std::chrono::microseconds>(realEnd - start);
    cout << "Complete process took " << (timePassed.count() / 1000) << "ms\n";

    return 0;
}

// Leah - how to run in command line:
// compile: g++ -o rasterize.exe rasterize.cpp filehandler.cpp barycentric.cpp
// execute: read.exe [fileName no extention]