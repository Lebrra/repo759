#include "hip/hip_runtime.h"
// inspiration: https://github.com/ssloy/tinyrenderer/wiki/Lesson-2:-Triangle-rasterization-and-back-face-culling

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <string>
#include "pixel.cuh"
#include "sizeAdjuster.cuh"
#include "filehandler.cuh"

using namespace std;

// hardset output height and width
const int definedSize = 256;
const float padding = 10;

void readyOutputFile(string fileName, long time){
    ofstream of;
    of.open(fileName + "_output.txt", ofstream::out | ofstream::trunc);
    of << time << endl;
    of.close();
}

void writeVertex(string fileName, float* vertex){
    fstream f;
    f.open(fileName + "_output.txt", ios::app);
    f << vertex[0] << ", " << vertex[1] << ", " << vertex[2] << endl;
    f.close();
}

int main(int argc, char** argv) {
    if (argc <= 1){
        cout << "Please include a filename (excluding the '_vertices.txt' or '_faces.txt') to rasterize!";
        return 0;
    }
    // else I could add optional size values, but let's just hardset it to 256...
    if (definedSize > 1024 || definedSize <= 0){
        cout << "Invalid image size; please pick a value between 1 and 1024";
        return 0;
    }

    auto start = chrono::steady_clock::now();

    // seeding with chrono so I don't have to include the ctime library:
	srand(chrono::system_clock::now().time_since_epoch().count());

    string fileName = argv[1];
    cout << "Processing file: " << fileName << endl;

    // gather data:
    cout << "Getting vertex count..." << endl;
    int vertCount = getVertexCount(fileName);
    cout << "Getting face count..." << endl;
    int triangleCount = getFaceCount(fileName);

    cout << "Alotting pointers..." << endl;
    float* vertices = (float*)malloc(sizeof(float) * vertCount * 3);
    int* faces = (int*)malloc(sizeof(int) * triangleCount * 3);
    float *dVerts;
    
    cout << "Reading all vertices..." << endl;
    readVertices(fileName, vertices);
    cout << "Reading all faces..." << endl;
    readFaces(fileName, faces);

    cout << "This file has " << vertCount << " vertices and " << triangleCount << " triangles!" << endl;

    cout << "Adjusting to size..." << endl;
    hipMalloc((void**)&dVerts, sizeof(float) * vertCount*3);
    hipMemcpy(dVerts, &vertices, sizeof(float) * vertCount*3, hipMemcpyHostToDevice);

    adjustSize(dVerts, vertCount, definedSize, padding);
    hipMemcpy(&vertices, dVerts, sizeof(float) * vertCount*3, hipMemcpyDeviceToHost);
    hipFree(dVerts);

    int pointTests[definedSize * definedSize], *dPoints;
    hipMalloc((void**)&dPoints, sizeof(int) * definedSize * definedSize);
    hipMemset(dPoints, 0, sizeof(int) * definedSize * definedSize);

    // do math
    int validTriangles = 0;
    cout << "Comparing pixels with triangles..." << endl;
    float triangle[6], *dTri;
    hipMalloc((void**)&dTri, sizeof(float) * 6);
    for(int tri = 0; tri < triangleCount; tri++){
        int face1 = faces[tri * 3];
        int face2 = faces[tri * 3 + 1];
        int face3 = faces[tri * 3 + 2];

        triangle[0] = vertices[face1*3];
        triangle[1] = vertices[face1*3 + 1];
        triangle[2] = vertices[face2*3];
        triangle[3] = vertices[face2*3 + 1];
        triangle[4] = vertices[face3*3];
        triangle[5] = vertices[face3*3 + 1];

        // validate triangle: (if two points are only different in the z direction then let's just skip it)
        if ((triangle[0] == triangle[2] && triangle[1] == triangle[3]) ||
            (triangle[0] == triangle[4] && triangle[1] == triangle[5]) ||
            (triangle[2] == triangle[4] && triangle[3] == triangle[5])){
                continue;
            }
        validTriangles++;
        hipMemcpy(dTri, &triangle, sizeof(float) * 6, hipMemcpyHostToDevice);

        // do parallelism here
        inTriangle<<<definedSize, definedSize>>>(dTri, dPoints, validTriangles);
    }
    hipFree(dTri);

    hipMemcpy(&pointTests, dPoints, sizeof(int) * definedSize * definedSize, hipMemcpyDeviceToHost);
    hipFree(dPoints);

    // create random colors:
    cout << "Generating triangle colors...\n";
    float* colors = (float*)malloc(sizeof(float) * 3 * validTriangles);
    for(int i = 0; i < validTriangles * 3; i++){
        colors[i] = static_cast <float> (rand() / static_cast <float> (RAND_MAX));
    }

    auto end = chrono::steady_clock::now();
	auto timePassed = chrono::duration_cast<std::chrono::microseconds>(end - start);

    // write
    cout << "Rasterization completed after " << (timePassed.count() / 1000) << "ms!\nWriting results...\n";
    float* pixel = (float*)malloc(sizeof(float) * 3);
    readyOutputFile(fileName, (timePassed.count() / 1000));
    for (int i = 0; i < definedSize * definedSize; i++){
        if (pointTests[i] <= 0) {
            pixel[0] = pixel[1] = pixel[2] = 0;
        }
        else {
            // do color
            pixel[0] = colors[(pointTests[i] - 1)*3];
            pixel[1] = colors[(pointTests[i] - 1)*3 + 1];
            pixel[2] = colors[(pointTests[i] - 1)*3 + 2];
        }
        writeVertex(fileName, pixel);
    }
    free(pixel);

    cout << "All done!" << endl;
    auto realEnd = chrono::steady_clock::now();
	timePassed = chrono::duration_cast<std::chrono::microseconds>(realEnd - start);
    cout << "Complete process took " << (timePassed.count() / 1000) << "ms\n";

    return 0;
}

// Leah - how to run in command line:
// compile: g++ -o rasterize.exe rasterize.cpp filehandler.cpp barycentric.cpp
// execute: read.exe [fileName no extention]