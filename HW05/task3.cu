#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

__global__ void arrayInit(float *a, float r, int n){
    int index = threadIdx.x + blockIdx.x * 512;
    if (index < n) {
        a[index] = r;
    }
}

int main(int argc, char* argv[]) {
    int n = 16;
    float hA[n], *dA;

    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, 0, n * sizeof(float));
    arrayInit<<<2, 8>>>(dA, 5., n);

    //random_device entropy_source;
    //mt19937 generator(entropy_source());
    //uniform_real_distribution<float> dist(0, 100);
    //auto r = dist(generator);

    vscale<<<2, 8>>>(dA, dA, n);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}