#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

__global__ void vscaleInt(const float *a, float *b, unsigned int n){
    int index = threadIdx.x + blockIdx.x * 512;
    printf("index = %d | n = %d\n", index, n);
    if (index < n) {
        printf("a = %f | b = %f | a*b = %f \n", a[index], b[index], a[index] * b[index]);
        b[index] *= a[index];
    }
}

int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);

    float hB[n], *dB, *dA;

    random_device entropy_source;
    mt19937 generator(entropy_source());
    uniform_real_distribution<float> distA(0., 20.);
    uniform_real_distribution<float> distB(0., 1.);

    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, distA(generator), n * sizeof(float));
    hipMalloc((void**)&dB, sizeof(float) * n);
    hipMemset(dB, distB(generator), n * sizeof(float));

    int threads = 512;
    int blocks = (n + threads - 1) / threads;
    printf("threads = %d | blocks = %d\n", threads, blocks);

    vscaleInt<<<blocks, threads>>>(dA, dB, n);
    hipDeviceSynchronize();

    hipMemcpy(&hB, dB, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < 5; i++) {
        cout << " dB = " << hB[i] << endl;
    }
    cout << endl;

    hipFree(dB);
    return 0;
}