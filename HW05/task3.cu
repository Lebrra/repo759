#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);

    float hB[n], *dB, *dA;

    random_device entropy_source;
    mt19937 generator(entropy_source());
    uniform_real_distribution<float> distA(0., 20.);
    uniform_real_distribution<float> distB(0., 1.);

    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, 0, n * sizeof(float));
    hipMalloc((void**)&dB, sizeof(float) * n);
    hipMemset(dB, 0, n * sizeof(float));
    
    // set dA and dB to random values:
    for(int i = 0; i < n; i++){
        dA[i] = distA(generator);
        dB[i] = distB(generator);
    }

    int t = 512;
    int b = (n + t - 1) / t;
    printf("threads = %d | blocks = %d\n", t, b);

    vscale<<<b, t>>>(dA, dB, n);
    hipDeviceSynchronize();

    hipMemcpy(&hB, dB, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < 5; i++) {
        cout << " hB = " << hB[i] << endl;
    }
    cout << endl;

    hipFree(dB);
    return 0;
}