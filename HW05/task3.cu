#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

int main(int argc, char* argv[]) {
    int n = 16;
    float hA[n], *dA;

    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, 5., n * sizeof(float));

    random_device entropy_source;
    mt19937 generator(entropy_source());
    uniform_real_distribution<float> dist(0, 100);
    auto r = dist(generator);

    vscale<<<2, 8>>>(dA, dA, n);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}