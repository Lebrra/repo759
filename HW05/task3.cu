#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

int main(int argc, char* argv[]) {
    int n = 16;
    float hA[n], *dA;

    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, 0, n * sizeof(float));
    for(int i = 0; i < n; i++){
        //hipMemset(dA[i], 5., sizeof(float));
        dA[i] = 5;
    }

    //random_device entropy_source;
    //mt19937 generator(entropy_source());
    //uniform_real_distribution<float> dist(0, 100);
    //auto r = dist(generator);

    vscale<<<2, 8>>>(dA, dA, n);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}