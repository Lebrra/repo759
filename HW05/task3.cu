#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
using namespace std;

__global__ void arrayInit(float *a, int n, float min, float max){
    int index = threadIdx.x + blockIdx.x * 512;
    if (index < n) {
        //random_device entropy_source;
        mt19937 generator(index);
        uniform_real_distribution<float> dist(min, max);

        if (index == 0){
            for(int i = 0; i < 10; i++){
                float rand = dist(generator);
                printf("%d = %f\n", i, rand);
            }
        }

        a[index] = dist(generator);
    }
}

int main(int argc, char* argv[]) {
    int n = atoi(argv[1]);
    float hB[n], *dB, *dA;

    // prepping threads and blocks:
    int t = 512;
    int b = (n + t - 1) / t;
    printf("threads = %d | blocks = %d\n", t, b);

    // randomization:
    //random_device entropy_source;
    //mt19937 generator(entropy_source());
    //uniform_real_distribution<float> distA(0., 20.);
    //uniform_real_distribution<float> distB(0., 1.);

    // array initialization:
    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMemset(dA, 0, n * sizeof(float));
    hipMalloc((void**)&dB, sizeof(float) * n);
    hipMemset(dB, 0, n * sizeof(float));
    arrayInit<<<b, t>>>(dA, n, 0., 20.);
    arrayInit<<<b, t>>>(dB, n, 0., 1.);
    hipDeviceSynchronize();

    // do math:
    vscale<<<b, t>>>(dA, dB, n);
    hipDeviceSynchronize();

    // results:
    hipMemcpy(&hB, dB, sizeof(float) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < 5; i++) {
        cout << " hB = " << hB[i] << endl;
    }
    cout << endl;

    hipFree(dB);
    hipFree(dA);
    return 0;
}