#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
using namespace std;

__global__ void algebraKernel(int* dA, int a) { 
    printf("a = %d", a);
    int index = threadIdx.x + blockIdx.x * 8;
    dA[index] = threadIdx.x * a + blockIdx.x;
}

int main() {
    int n = 16;
    int hA[n], *dA;
    //allocate memory on the device (GPU); zero out all entries in this device array
    hipMalloc((void**)&dA, sizeof(int) * n);
    hipMemset(dA, 0, n * sizeof(int));

    mt19937 generator(12354);
    uniform_int_distribution<int> dist(0., 100.);
    auto r = dist(generator);

    algebraKernel<<<2, 8>>>(dA, r);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(int) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}