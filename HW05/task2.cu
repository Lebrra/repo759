#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void algebraKernel(int* dA, int a) { 
    printf("a = %d", a);
    int index = threadIdx.x + blockIdx.x * 8;
    dA[index] = threadIdx.x * a + blockIdx.x;
}

int main() {
    int n = 16;
    int hA[n], *dA;
    //allocate memory on the device (GPU); zero out all entries in this device array
    hipMalloc((void**)&dA, sizeof(int) * n);
    hipMemset(dA, 0, n * sizeof(int));

    float r = static_cast <int> (rand() / static_cast <int> (RAND_MAX / 100)) - 50;

    algebraKernel<<<2, 8>>>(dA, r);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(int) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}