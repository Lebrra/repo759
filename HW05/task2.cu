#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
using namespace std;

__global__ void algebraKernel(int* dA, int a) { 
    int index = threadIdx.x + blockIdx.x * 8;
    dA[index] = threadIdx.x * a + blockIdx.x;
}

int main() {
    int n = 16;
    int hA[n], *dA;

    hipMalloc((void**)&dA, sizeof(int) * n);
    hipMemset(dA, 0, n * sizeof(int));

    random_device entropy_source;
    mt19937 generator(entropy_source());
    uniform_int_distribution<int> dist(0, 100);
    auto r = dist(generator);

    algebraKernel<<<2, 8>>>(dA, r);
    hipDeviceSynchronize();

    hipMemcpy(&hA, dA, sizeof(int) * n, hipMemcpyDeviceToHost);

    cout << "Results: " << endl;
    for (int i = 0; i < n; i++) cout << hA[i] << " ";
    cout << endl;

    hipFree(dA);
    return 0;
}