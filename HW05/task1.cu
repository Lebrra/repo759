#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void factorialKernel(int a) { 
    int b = 1;
    for (int i = a; i > 1; i++){
        b *= i;
    }
    printf("%d! = %d\n", a, b);
}

int main() {
    printf("printing...\n");
    for (int i = 1; i <= 8; i++){
        factorialKernel<<<1, 8>>>(i);
    }
    hipDeviceSynchronize();
    return 0;
}