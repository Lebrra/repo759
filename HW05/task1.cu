#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void factorialKernel() { 
    int a = threadIdx.x + 1;
    int b = 1;
    for (int i = 2; i <= a; i++) {
        b *= i;
    }
    printf("%d! = %d\n", a, b);
}

int main() {
    factorialKernel<<<1, 8>>>();
    hipDeviceSynchronize();
    return 0;
}