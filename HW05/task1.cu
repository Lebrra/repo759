#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;

__global__ void factorialKernel(int a) { 
    int b = 1;
    for (int i = a; i > 1; i++){
        b *= i;
    }
    printf(b);
    //printf("%d! = %d", a, b);
}

int main() {
    printf("printing...\n")
    for (int i = 1; i <= 8; i++){
        factorialKernel<<<1, 8>>>(i);
    }
    hipDeviceSynchronize();
    return 0;
}