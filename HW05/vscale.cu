#include <hip/hip_runtime.h>

#ifndef VSCALE_CUH
__global__ void vscale(const float *a, float *b, unsigned int n){
    int index = threadIdx.x + blockIdx.x * 512;
    printf("index = %d | n = %d\n", index, n);
    if (index < n) {
        printf("a = %f | b = %f | a*b = %f \n", a[index], b[index], a[index] * b[index]);
        b[index] *= a[index];
    }
}

#endif